#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/complex.h>

#define SIG_LEN 81920
#define N_SIGS (20*6*3)

/*int main() {
    hipfftHandle plan;
    hipfftComplex *data;
    float time, cumulative_time = 0.f;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    hipMalloc((void**)&data, sizeof(hipfftComplex)*NX*BATCH);
    if (hipGetLastError() != hipSuccess){
        fprintf(stderr, "Cuda error: Failed to allocate\n");
        return 0;
    }

    if (hipfftPlan1d(&plan, NX, HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS){
        fprintf(stderr, "CUFFT error: Plan creation failed");
        return 0;
    }

    for (int i=0; i<20; i++) {
        hipEventRecord(start, 0);
        if (hipfftPlan1d(&plan, NX, HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT error: Plan creation failed");
            return 0;
        }
        // Note: * Identical pointers to input and output arrays implies in-place transformation
        if (hipfftExecC2C(plan, data, data, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
            return 0;
        }

        if (hipfftExecC2C(plan, data, data, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT error: ExecC2C Inverse failed");
            return 0;  }
        //Results may not be immediately available so block device until all * tasks have completed
        if (hipDeviceSynchronize() != hipSuccess){
            fprintf(stderr, "Cuda error: Failed to synchronize\n");
            return 0;
        }
        hipfftDestroy(plan);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        cumulative_time = cumulative_time + time;
    }


    printf("FFT + IFFT time:  %3.5f ms \n", cumulative_time);


      // Divide by number of elements in data set to get back original data
    hipfftDestroy(plan);
    hipFree(data);
    return 0;
}*/
__global__ void multiply(hipFloatComplex* samples,hipFloatComplex* filter,
                            hipFloatComplex* result, int num_samps) {
    auto channel_off = num_samps * gridDim.y;
    auto samp_off = blockIdx.x * blockDim.x + threadIdx.x;

    hipFloatComplex res;
    res.x = samples[channel_off + samp_off].x * filter[samp_off].x - samples[channel_off + samp_off].y * filter[samp_off].y;
    res.y = samples[channel_off + samp_off].x * filter[samp_off].y + samples[channel_off + samp_off].y * filter[samp_off].x;

    result[channel_off + samp_off] = res;

}




struct ElementWiseProductBasic : public thrust::binary_function<hipFloatComplex,hipFloatComplex,hipFloatComplex>
{
    __host__ __device__
    hipFloatComplex operator()(const hipFloatComplex& v1, const hipFloatComplex& v2) const
    {
        hipFloatComplex res;
        res.x = v1.x * v2.x - v1.y * v2.y;
        res.y = v1.x * v2.y + v1.y * v2.x;
        return res;
    }
};
void throw_on_cuda_error(hipError_t code, const char *file, int line)
{
  if(code != hipSuccess)
  {
    std::stringstream ss;
    ss << file << "(" << line << ")";
    std::string file_and_line;
    ss >> file_and_line;
    throw thrust::system_error(code, thrust::cuda_category(), file_and_line);
  }
}
int main(){
  int rank = 1;                           // --- 1D FFTs
  int n[] = { SIG_LEN };                 // --- Size of the Fourier transform
  int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
  int idist = SIG_LEN, odist = (SIG_LEN); // --- Distance between batches
  int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
  int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
  int batch = N_SIGS;                      // --- Number of batched executions
  hipfftHandle plan;
  auto res = hipfftPlanMany(&plan, rank, n,
                inembed, istride, idist,
                onembed, ostride, odist, HIPFFT_C2C, batch);
  if (res != HIPFFT_SUCCESS) {printf("plan create fail\n"); return 1;}

  //hipFloatComplex *h_signal, *d_signal, *h_result, *d_result;
  hipEvent_t start, stop;
  float time = 0.0;


  hipEventCreate(&start);
  hipEventCreate(&stop);

  auto h_signal = thrust::host_vector<hipFloatComplex>(N_SIGS * SIG_LEN);
  for (int i = 0; i < N_SIGS; i ++)
    for (int j = 0; j < SIG_LEN; j++)
      h_signal[(i*SIG_LEN) + j] = make_hipFloatComplex(sin((i+1)*6.283*j/SIG_LEN), 0);


  //hipMalloc(&d_signal, N_SIGS*SIG_LEN*sizeof(hipFloatComplex));
  //hipMalloc(&d_result, N_SIGS*SIG_LEN*sizeof(hipFloatComplex));
  hipEventRecord(start, 0);
  thrust::device_vector<hipFloatComplex> d_signal = h_signal;
  thrust::device_vector<hipFloatComplex> d_result(N_SIGS * SIG_LEN);
  thrust::device_vector<hipFloatComplex> d_result_modified(N_SIGS * SIG_LEN);
  thrust::device_vector<hipFloatComplex> d_filter(N_SIGS * SIG_LEN);
  //hipMemcpy(d_signal, h_signal, N_SIGS*SIG_LEN*sizeof(hipFloatComplex), hipMemcpyHostToDevice);

  res = hipfftPlanMany(&plan, rank, n,
                inembed, istride, idist,
                onembed, ostride, odist, HIPFFT_C2C, batch);
  if (res != HIPFFT_SUCCESS) {printf("plan create fail\n"); return 1;}

  //FFT
  auto d_signal_p = thrust::raw_pointer_cast(d_signal.data());
  auto d_result_p = thrust::raw_pointer_cast(d_result.data());
  auto d_result_modified_p = thrust::raw_pointer_cast(d_result_modified.data());
  res = hipfftExecC2C(plan, d_signal_p, d_result_p, HIPFFT_FORWARD);
  if (res != HIPFFT_SUCCESS) {printf("forward transform fail\n"); return 1;}
  //hipMemcpy(h_result, d_result, N_SIGS*SIG_LEN*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
/*  thrust::transform(d_result.begin(), d_result.end(),
                          d_filter.begin(), d_result_modified.begin(),
                          ElementWiseProductBasic());*/
  dim3 dimGrid(SIG_LEN/1024,N_SIGS,1);
  dim3 dimBlock(1024);
  //multiply<<<dimGrid,dimBlock>>>(d_signal_p,d_result_p,d_result_modified_p,SIG_LEN);
  //throw_on_cuda_error(hipPeekAtLastError(), __FILE__,__LINE__);

  //IFFT
  res = hipfftExecC2C(plan, d_signal_p, d_result_p, HIPFFT_BACKWARD);
  if (res != HIPFFT_SUCCESS) {printf("forward transform fail\n"); return 1;}
  //hipMemcpy(h_result, d_result, N_SIGS*SIG_LEN*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
  auto h_result = d_result;
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  printf("FFT/IFFT time:  %3.5f ms \n", time);

  for (int i = 0; i < N_SIGS; i++){
    for (int j = 0; j < 10; j++)
      printf("%.3f ", hipCrealf(h_result[(i*SIG_LEN)+j]));
    printf("\n"); }

  return 0;
}